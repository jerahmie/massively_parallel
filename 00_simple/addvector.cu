#include <iostream>
#include <hip/hip_runtime.h>

__global__ void VecAdd(float* A, float* B, float* C, int n)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main()
{
    std::cout << "Hello World.\n";
}