#include <stdio.h>
#include "hip/hip_runtime.h"

__global__ void hello_gpu_device() {
	printf("Hello World from GPU!\n");
}

extern "C" {
	void hello_gpu(void) {
		hello_gpu_device<<<1,1>>>();
	}
}
