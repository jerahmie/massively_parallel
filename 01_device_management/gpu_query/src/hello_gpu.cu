#include <stdio.h>

#include "hip/hip_runtime.h"

__host__ void hello_gpu_device() {
    printf("Hello World from GPU!\n");
}

extern "C" {
    void hello_gpu(void) {
        hello_gpu_device();
    }

    int print_device_properties(hipDeviceProp_t *pdevprop) {
        printf("CUDA Device Properties: \n\n");
        printf(" name: %s\n", pdevprop->name);
        printf(" uuid: %02X\n", pdevprop->uuid);
        printf(" Total global memory : %lu \n", pdevprop->totalGlobalMem);
        printf(" Shared memory per block: %lu\n", pdevprop->sharedMemPerBlock);
        printf(" Registers per block: %d\n", pdevprop->regsPerBlock);
        printf(" Warp Size: %d\n", pdevprop->warpSize);
        printf(" Memory Pitch: %lu\n", pdevprop->memPitch);
        printf(" Max Threads per Block: %d\n", pdevprop->maxThreadsPerBlock);
        printf(" Max Threads dimension: %d, %d, %d\n", pdevprop->maxThreadsDim[0], pdevprop->maxThreadsDim[1], pdevprop->maxThreadsDim[2]);
        printf(" Max Grid Size: %d, %d, %d\n", pdevprop->maxGridSize[0], pdevprop->maxGridSize[1], pdevprop->maxGridSize[2]);
        printf(" Clock Rage: %d\n", pdevprop->clockRate);
        printf(" Total Constant Memory: %lu\n", pdevprop->totalConstMem);
        printf(" Major: %d\n", pdevprop->major);
        printf(" Minor: %d\n", pdevprop->minor);
        printf(" Texture Alignment: %lu\n", pdevprop->textureAlignment);
        printf(" Texture Pitch Alignment: %lu\n", pdevprop->texturePitchAlignment);
        printf(" Device Overlap: %d\n", pdevprop->deviceOverlap);
        printf(" Mulitprocessor Count: %d\n", pdevprop->multiProcessorCount);
        printf(" Kernel Exec Timeout Enabled: %d\n", pdevprop->kernelExecTimeoutEnabled);
        printf(" Integrated: %d\n", pdevprop->integrated);
        printf(" Can Map Host Memory: %d\n", pdevprop->canMapHostMemory);
        printf(" Compute Mode: %d\n", pdevprop->computeMode);
        printf(" Max Texture 1D: %d\n", pdevprop->maxTexture1D);
        printf(" Max Texture 1D Mipmap: %d\n", pdevprop->maxTexture1DMipmap);
        printf(" Max Surface 1D Linear: %d\n", pdevprop->maxTexture1DLinear);

        return 0;
    }

    int device_properties(int device_id) {
        int  ret_val = 0;
        hipDeviceProp_t dev_properties;
        ret_val = hipGetDeviceProperties(&dev_properties, device_id);
        print_device_properties(&dev_properties);

        return ret_val;
    }
	

    int query_gpu(){
        hipInit(0);
        int device = -1;
        int deviceCount = 0;
        hipGetDeviceCount(&deviceCount);
        hipGetDevice(&device);
        device_properties(device);

        return deviceCount;
    }
}

