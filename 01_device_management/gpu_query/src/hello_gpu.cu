#include <stdio.h>
#include "hip/hip_runtime.h"

__host__ void hello_gpu_device() {
	printf("Hello World from GPU!\n");
}

extern "C" {
	void hello_gpu(void) {
		hello_gpu_device();
	}

	int query_gpu(){
		hipInit(0);
		int device = -1;
		int deviceCount = 0;
		hipGetDeviceCount(&deviceCount);
		hipGetDevice(&device);
		return deviceCount;

	}
}

