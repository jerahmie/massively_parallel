#include <stdio.h>

#include "hip/hip_runtime.h"

__host__ void hello_gpu_device() {
    printf("Hello World from GPU!\n");
}

extern "C" {
    void hello_gpu(void) {
        hello_gpu_device();
    }
    
    int print_device_properties(hipDeviceProp_t *pdevprop) {
        printf("CUDA Device Properties: \n\n");
        printf(" name: %s\n", pdevprop->name);
	printf(" uuid: %x\n", pdevprop->uuid);
        printf(" Total global memory : %lu \n", pdevprop->totalGlobalMem);
        printf(" Shared memory per block: %lu\n", pdevprop->sharedMemPerBlock);
        printf(" Registers per block: %d\n", pdevprop->regsPerBlock);
        printf(" Warp Size: %d\n", pdevprop->warpSize);
        printf(" Memory Pitch: %lu\n", pdevprop->memPitch);
        printf(" Max Threads per Block: %d\n", pdevprop->maxThreadsPerBlock);
        printf(" Max Threads dimension: %d, %d, %d\n", pdevprop->maxThreadsDim[0], pdevprop->maxThreadsDim[1], pdevprop->maxThreadsDim[2]);
        printf(" Max Grid Size: %d, %d, %d\n", pdevprop->maxGridSize[0], pdevprop->maxGridSize[1], pdevprop->maxGridSize[2]);
        printf(" Clock Rage: %d\n", pdevprop->clockRate);
        printf(" Total Constant Memory: %lu\n", pdevprop->totalConstMem);
        printf(" Major: %d\n", pdevprop->major);
        printf(" Minor: %d\n", pdevprop->minor);
        printf(" Texture Alignment: %lu\n", pdevprop->textureAlignment);
        printf(" Texture Pitch Alignment: %lu\n", pdevprop->texturePitchAlignment);
        printf(" Device Overlap: %d\n", pdevprop->deviceOverlap);
        printf(" Mulitprocessor Count: %d\n", pdevprop->multiProcessorCount);
        printf(" Kernel Exec Timeout Enabled: %d\n", pdevprop->kernelExecTimeoutEnabled);
        printf(" Integrated: %d\n", pdevprop->integrated);
        printf(" Can Map Host Memory: %d\n", pdevprop->canMapHostMemory);
        printf(" Compute Mode: %d\n", pdevprop->computeMode);
        printf(" Max Texture 1D: %d\n", pdevprop->maxTexture1D);
        printf(" Max Texture 1D Mipmap: %d\n", pdevprop->maxTexture1DMipmap);
        printf(" Max Surface 1D Linear: %d\n", pdevprop->maxTexture1DLinear);
        printf(" Max Texture 2D: %d, %d\n", pdevprop->maxTexture2D[0],
                                            pdevprop->maxTexture2D[1]);
        printf(" Max Texture 2D Mipmap: %d, %d\n", pdevprop->maxTexture2DMipmap[0],
                                                   pdevprop->maxTexture2DMipmap[1]);
        printf(" Max Texture 2D Linear: %d, %d, %d\n", pdevprop->maxTexture2DLinear[0],
                                                       pdevprop->maxTexture2DLinear[1],
                                                       pdevprop->maxTexture2DLinear[2]);
        printf(" Max Texture 2D Gather: %d, %d\n", pdevprop->maxTexture2DGather[0],
                                                   pdevprop->maxTexture2DGather[1]);
        printf(" Max Texture 3D: %d, %d, %d\n", pdevprop->maxTexture3D[0],
                                                pdevprop->maxTexture3D[1],
                                                pdevprop->maxTexture3D[2]);
        printf(" Max Texure 3D Alt: %d, %d, %d\n", pdevprop->maxTexture3DAlt[0],
                                                   pdevprop->maxTexture3DAlt[1],
                                                   pdevprop->maxTexture3DAlt[2]);
        printf(" Max Texture Cubemap: %d\n", pdevprop->maxTextureCubemap);
        printf(" Max Texture 1D Layered: %d, %d\n", pdevprop->maxTexture1DLayered[0],
                                                    pdevprop->maxTexture1DLayered[1]);
        printf(" Max Texture 2D Layered: %d, %d, %d\n", pdevprop->maxTexture2DLayered[0],
                                                        pdevprop->maxTexture2DLayered[1],
                                                        pdevprop->maxTexture2DLayered[2]);
        printf(" Max Texture Cubemap Layered: %d, %d\n", pdevprop->maxTextureCubemapLayered[0],
                                                         pdevprop->maxTextureCubemapLayered[1]);
        printf(" Max Surface 1D: %d\n", pdevprop->maxSurface1D);
        printf(" Max Surface 2D: %d, %d\n", pdevprop->maxSurface2D[0],
                                            pdevprop->maxSurface2D[1]);
        printf(" Max Surface 3D: %d, %d, %d\n", pdevprop->maxSurface3D[0],
                                                pdevprop->maxSurface3D[1],
                                                pdevprop->maxSurface3D[2]);
        printf(" Max Surface 1D Layered: %d, %d\n", pdevprop->maxSurface1DLayered[0],
                                                    pdevprop->maxSurface1DLayered[1]);
        printf(" Max Surface 2D Layered: %d, %d, %d\n", pdevprop->maxSurface2DLayered[0],
                                                        pdevprop->maxSurface2DLayered[1],
                                                        pdevprop->maxSurface2DLayered[2]);
        printf(" Max Surface Cubemap: %d\n", pdevprop->maxSurfaceCubemap);
        printf(" Max Surface Cubemap Layered: %d, %d\n", pdevprop->maxSurfaceCubemapLayered[0],
                                                         pdevprop->maxSurfaceCubemapLayered[1]);
        printf(" Surface Alignment: %lu\n", pdevprop->surfaceAlignment);
        printf(" Concurrent Kernels: %d\n", pdevprop->concurrentKernels);
        printf(" ECC Enabled: %d\n", pdevprop->ECCEnabled);
        printf(" PCI Bus ID: %d\n", pdevprop->pciBusID);
        printf(" PCI Device ID: %d\n", pdevprop->pciDeviceID);
        printf(" PCI Domain ID: %d\n", pdevprop->pciDomainID);
        printf(" TCC Driver: %d\n", pdevprop->tccDriver);
        printf(" Async Engine Count: %d\n", pdevprop->asyncEngineCount);
        printf(" Unified Addressing: %d\n", pdevprop->unifiedAddressing);
        printf(" Memory Clock Rate: %d\n", pdevprop->memoryClockRate);
        printf(" Memory Bus Width: %d\n", pdevprop->memoryBusWidth);
        printf(" L2 Cache Size: %d\n", pdevprop->l2CacheSize);
        printf(" Persisting L2 Cache Max Size: %d\n", pdevprop->persistingL2CacheMaxSize);
        printf(" Max Thread Per Multiprocessor: %d\n", pdevprop->maxThreadsPerMultiProcessor);
        printf(" Stream Priorities Supported: %d\n", pdevprop->streamPrioritiesSupported);
        printf(" Global L1 Cache Supported: %d\n", pdevprop->globalL1CacheSupported);
        printf(" Local L1 Cache Supported: %d\n", pdevprop->localL1CacheSupported);
        printf(" Shared Mem Per Multiprocessor: %lu\n", pdevprop->sharedMemPerMultiprocessor);
        printf(" Registers Per Multiprocessor: %d\n", pdevprop->regsPerMultiprocessor);
        printf(" Managed Memory: %d\n", pdevprop->managedMemory);
        printf(" Is Multi GPU Board: %d\n", pdevprop->isMultiGpuBoard);
        printf(" Multi-GPU Board Group ID: %d\n", pdevprop->multiGpuBoardGroupID);
        printf(" Single to Double Precision Perf Ratio: %d\n", pdevprop->singleToDoublePrecisionPerfRatio);
        printf(" Pageable Memory Access: %d\n", pdevprop->pageableMemoryAccess);
        printf(" Concurrent Managed Access: %d\n", pdevprop->concurrentManagedAccess);
        printf(" Compute Preemption Supported: %d\n", pdevprop->computePreemptionSupported);
        printf(" Can Use Host Pointer for Registered Memory: %d\n", pdevprop->canUseHostPointerForRegisteredMem);
        printf(" Cooperative Launch: %d\n", pdevprop->cooperativeLaunch);
        printf(" Cooperative Multi-Device Launch %d\n", pdevprop->cooperativeMultiDeviceLaunch);
        printf(" Pageable Memory access Uses Host Page Tables: %d\n", pdevprop->pageableMemoryAccessUsesHostPageTables);
        printf(" Direct Managed Memory Access From Host: %d\n", pdevprop->directManagedMemAccessFromHost);
        printf(" Access Policy Max Windows Size: %d\n", pdevprop->accessPolicyMaxWindowSize);

        return 0;
    }

    int device_properties(int device_id) {
        int  ret_val = 0;
        hipDeviceProp_t dev_properties;
        ret_val = hipGetDeviceProperties(&dev_properties, device_id);
        print_device_properties(&dev_properties);

        return ret_val;
    }
	

    int query_gpu(){
        hipInit(0);
        int device = -1;
        int deviceCount = 0;
        hipGetDeviceCount(&deviceCount);
        hipGetDevice(&device);
        device_properties(device);

        return deviceCount;
    }
}

